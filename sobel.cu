#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <chrono>
using namespace std;
using namespace cv;

__global__ void grayscale(unsigned char* mA_inter,unsigned char* rgb){
	int x_current = blockDim.x * blockIdx.x + threadIdx.x;
	mA_inter[x_current] = (307*rgb[3*x_current]+604*rgb[3*x_current+1]+113*rgb[3*x_current+2])/1024;
}

__global__ void sobel(unsigned char* mA_d,unsigned char* mA_inter,int width,int height){
	int x_current = blockDim.x * blockIdx.x + threadIdx.x;
	if (
		x_current>=width &&
		x_current<=(height-1)*width &&
		x_current%width!=0 &&
		x_current%width!=(width-1)
	){
		float h = mA_inter[x_current-1-width] - mA_inter[x_current+1-width]
				+ 2*mA_inter[x_current-1] - 2*mA_inter[x_current+1]
				+ mA_inter[x_current-1+width] - mA_inter[x_current+1+width];
		float v = mA_inter[x_current-1-width] - mA_inter[x_current-1+width]
				+ 2*mA_inter[x_current-width] - 2*mA_inter[x_current+width]
				+ mA_inter[x_current+1-width] - mA_inter[x_current+1+width];
		h = h > 255 ? 255 : h;
		v = v > 255 ? 255 : v;
		float res = h*h + v*v;
		res = res > 255*255 ? 255*255 : res;
		mA_d[x_current] = sqrt(res);
	}
}

int main(int argc, char *argv[]){
	hipEvent_t event_deb;
	hipEvent_t event_fin;
	
	hipError_t cudaErrorIdentifier;
	
	float temps;
	cudaErrorIdentifier = hipEventCreate(&event_deb);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la création de l'événement event_deb"<<endl;
		
	cudaErrorIdentifier = hipEventCreate(&event_fin);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la création de l'événement event_fin"<<endl;

	cudaErrorIdentifier = hipEventRecord(event_deb,0);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur au lancement du record event_deb"<<endl;

	Mat m_in = imread(argv[1], IMREAD_UNCHANGED );
	auto rgb = m_in.data;
	
	vector< unsigned char > g( m_in.rows * m_in.cols);
	Mat m_out( m_in.rows, m_in.cols, CV_8UC1, g.data());
	
	unsigned char* mA_d=nullptr;
	unsigned char* mA_inter=nullptr;
	unsigned char* mA_rgb=nullptr;
	
	hipDeviceProp_t prop;
	cudaErrorIdentifier = hipGetDeviceProperties(&prop, 0);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la récupération des propriétés du device"<<endl;
	
	cudaErrorIdentifier = hipMalloc(&mA_d,g.size()*sizeof(unsigned char));
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à l'allocation de mA_d"<<endl;
	cudaErrorIdentifier = hipMalloc(&mA_inter,g.size()*sizeof(unsigned char));
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à l'allocation de mA_inter"<<endl;
	cudaErrorIdentifier = hipMalloc(&mA_rgb,m_in.rows*m_in.cols*3*sizeof(unsigned char));
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à l'allocation de mA_rgb"<<endl;
	
	cudaErrorIdentifier = hipMemcpy(mA_d,g.data(),g.size()*sizeof(unsigned char),hipMemcpyHostToDevice);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à l'envoi des données à mA_d"<<endl;
	cudaErrorIdentifier = hipMemcpy(mA_inter,g.data(),g.size()*sizeof(unsigned char),hipMemcpyHostToDevice);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à l'envoi des données à mA_inter"<<endl;
	cudaErrorIdentifier = hipMemcpy(mA_rgb,rgb,m_in.rows*m_in.cols*3*sizeof(unsigned char),hipMemcpyHostToDevice);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à l'envoi des données à mA_rgb"<<endl;
	
	int grid = 1;
	int block = m_in.rows * m_in.cols;
	while (block > prop.maxThreadsPerBlock){
		block -= prop.maxThreadsPerBlock;
		grid++;
	}
	if (grid > 1){
		block = prop.maxThreadsPerBlock;
	}
	grayscale<<<grid,block>>>(mA_inter,mA_rgb);
	cudaErrorIdentifier = hipGetLastError();
	if (cudaErrorIdentifier != hipSuccess){
		cout<<"Erreur à l'exécution de la fonction grayscale"<<endl;
		cout<<"détail : "<<hipGetErrorString(cudaErrorIdentifier)<<endl;
	}
	cudaErrorIdentifier = hipMemcpy(g.data(),mA_inter,g.size()*sizeof(unsigned char),hipMemcpyDeviceToHost);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la réception des données de mA_inter"<<endl;

	cudaErrorIdentifier = hipMemcpy(mA_inter,g.data(),g.size()*sizeof(unsigned char),hipMemcpyHostToDevice);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à l'envoi des données à mA_inter"<<endl;

	sobel<<<grid,block>>>(mA_d,mA_inter,m_in.cols,m_in.rows);
	cudaErrorIdentifier = hipGetLastError();
	if (cudaErrorIdentifier != hipSuccess){
		cout<<"Erreur à l'exécution de la fonction sobel"<<endl;
		cout<<"détail : "<<hipGetErrorString(cudaErrorIdentifier)<<endl;
	}
	cudaErrorIdentifier = hipMemcpy(g.data(),mA_d,g.size()*sizeof(unsigned char),hipMemcpyDeviceToHost);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la réception des données de mA_d"<<endl;

	std::string path(argv[1]);
	imwrite( "out_sobel_cu_"+path, m_out );
	
	cudaErrorIdentifier = hipEventRecord(event_fin,0);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur au lancement du record event_fin"<<endl;
		
	cudaErrorIdentifier = hipEventSynchronize(event_fin);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la synchronisation entre event_deb et event_fin"<<endl;
		
	cudaErrorIdentifier = hipEventElapsedTime(&temps,event_deb,event_fin);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur au calcul du temps entre event_deb et event_fin"<<endl;
		
	cout<<temps<<" millisecondes"<<endl;
	
	cudaErrorIdentifier = hipEventDestroy(event_deb);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la destruction de event_deb"<<endl;
		
	cudaErrorIdentifier = hipEventDestroy(event_fin);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la destruction de event_fin"<<endl;

	hipFree(mA_d);
	hipFree(mA_inter);
	hipFree(mA_rgb);
	return 0;
}




