#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <chrono>
using namespace std;
using namespace cv;

__global__ void grayscale(unsigned char* mA_inter,unsigned char* rgb,int width){
	auto i = blockIdx.y * blockDim.y + threadIdx.y;
	auto j = blockIdx.x * blockDim.x + threadIdx.x;
	mA_inter[i*width+j] = (307*rgb[3*(i*width+j)]+604*rgb[3*(i*width+j)+1]+113*rgb[3*(i*width+j)+2])/1024;
}

__global__ void sobel(unsigned char* mA_d,unsigned char* mA_inter,int width,int height){
	auto i = blockIdx.y * blockDim.y + threadIdx.y;
	auto j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i > 0 && i < height && j % width != 0 && j % width != width-1){
		float h = mA_inter[(i-1)*width+(j-1)] - mA_inter[(i-1)*width+(j+1)]
				+ 2*mA_inter[i*width+(j-1)] - 2*mA_inter[i*width+(j+1)]
				+ mA_inter[(i+1)*width+(j-1)] - mA_inter[(i+1)*width+(j+1)];
		float v = mA_inter[(i-1)*width+(j-1)] - mA_inter[(i+1)*width+(j-1)]
				+ 2*mA_inter[(i-1)*width+j] - 2*mA_inter[(i+1)*width+j]
				+ mA_inter[(i-1)*width+(j+1)] - mA_inter[(i+1)*width+(j+1)];
		h = h > 255 ? 255 : h;
		v = v > 255 ? 255 : v;
		float res = h*h + v*v;
		res = res > 255*255 ? 255*255 : res;
		mA_d[i*width+j] = sqrt(res);
	}
}

int main(int argc, char *argv[]){
	hipEvent_t event_deb;
	hipEvent_t event_fin;
	
	hipError_t cudaErrorIdentifier;
	
	float temps;
	cudaErrorIdentifier = hipEventCreate(&event_deb);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la création de l'événement event_deb"<<endl;
		
	cudaErrorIdentifier = hipEventCreate(&event_fin);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la création de l'événement event_fin"<<endl;

	cudaErrorIdentifier = hipEventRecord(event_deb,0);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur au lancement du record event_deb"<<endl;

	Mat m_in = imread(argv[1], IMREAD_UNCHANGED );
	auto rgb = m_in.data;
	
	vector< unsigned char > g( m_in.rows * m_in.cols);
	Mat m_out( m_in.rows, m_in.cols, CV_8UC1, g.data());
	
	unsigned char* mA_d=nullptr;
	unsigned char* mA_inter=nullptr;
	unsigned char* mA_rgb=nullptr;
	
	hipDeviceProp_t prop;
	cudaErrorIdentifier = hipGetDeviceProperties(&prop, 0);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la récupération des propriétés du device"<<endl;
	
	cudaErrorIdentifier = hipMalloc(&mA_d,g.size()*sizeof(unsigned char));
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à l'allocation de mA_d"<<endl;
	cudaErrorIdentifier = hipMalloc(&mA_inter,g.size()*sizeof(unsigned char));
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à l'allocation de mA_inter"<<endl;
	cudaErrorIdentifier = hipMalloc(&mA_rgb,m_in.rows*m_in.cols*3*sizeof(unsigned char));
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à l'allocation de mA_rgb"<<endl;
	
	cudaErrorIdentifier = hipMemcpy(mA_d,g.data(),g.size()*sizeof(unsigned char),hipMemcpyHostToDevice);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à l'envoi des données à mA_d"<<endl;
	cudaErrorIdentifier = hipMemcpy(mA_inter,g.data(),g.size()*sizeof(unsigned char),hipMemcpyHostToDevice);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à l'envoi des données à mA_inter"<<endl;
	cudaErrorIdentifier = hipMemcpy(mA_rgb,rgb,m_in.rows*m_in.cols*3*sizeof(unsigned char),hipMemcpyHostToDevice);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à l'envoi des données à mA_rgb"<<endl;
	
	/*int grid = 1;
	int block = m_in.rows * m_in.cols;
	while (block > prop.maxThreadsPerBlock){
		block -= prop.maxThreadsPerBlock;
		grid++;
	}
	if (grid > 1){
		block = prop.maxThreadsPerBlock;
	}*/
	dim3 block(sqrt(prop.maxThreadsPerBlock), sqrt(prop.maxThreadsPerBlock));
	dim3 grid((m_in.cols + block.x - 1) / block.x, (m_in.rows + block.y - 1) / block.y);
	grayscale<<<grid,block>>>(mA_inter,mA_rgb,m_in.cols);
	
	cudaErrorIdentifier = hipGetLastError();
	if (cudaErrorIdentifier != hipSuccess){
		cout<<"Erreur à l'exécution de la fonction grayscale"<<endl;
		cout<<"détail : "<<hipGetErrorString(cudaErrorIdentifier)<<endl;
	}
	cudaErrorIdentifier = hipMemcpy(g.data(),mA_inter,g.size()*sizeof(unsigned char),hipMemcpyDeviceToHost);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la réception des données de mA_inter"<<endl;

	cudaErrorIdentifier = hipMemcpy(mA_inter,g.data(),g.size()*sizeof(unsigned char),hipMemcpyHostToDevice);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à l'envoi des données à mA_inter"<<endl;

	sobel<<<grid,block>>>(mA_d,mA_inter,m_in.cols,m_in.rows);
	cudaErrorIdentifier = hipGetLastError();
	if (cudaErrorIdentifier != hipSuccess){
		cout<<"Erreur à l'exécution de la fonction sobel"<<endl;
		cout<<"détail : "<<hipGetErrorString(cudaErrorIdentifier)<<endl;
	}
	cudaErrorIdentifier = hipMemcpy(g.data(),mA_d,g.size()*sizeof(unsigned char),hipMemcpyDeviceToHost);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la réception des données de mA_d"<<endl;

	std::string path(argv[1]);
	imwrite( "out_sobel_cu_"+path, m_out );
	
	cudaErrorIdentifier = hipEventRecord(event_fin,0);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur au lancement du record event_fin"<<endl;
		
	cudaErrorIdentifier = hipEventSynchronize(event_fin);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la synchronisation entre event_deb et event_fin"<<endl;
		
	cudaErrorIdentifier = hipEventElapsedTime(&temps,event_deb,event_fin);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur au calcul du temps entre event_deb et event_fin"<<endl;
		
	cout<<temps<<" millisecondes"<<endl;
	
	cudaErrorIdentifier = hipEventDestroy(event_deb);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la destruction de event_deb"<<endl;
		
	cudaErrorIdentifier = hipEventDestroy(event_fin);
	if (cudaErrorIdentifier != hipSuccess)
		cout<<"Erreur à la destruction de event_fin"<<endl;

	hipFree(mA_d);
	hipFree(mA_inter);
	hipFree(mA_rgb);
	return 0;
}



